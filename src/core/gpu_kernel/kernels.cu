#include "hip/hip_runtime.h"
#include <hip/hip_cooperative_groups.h>
#include <thrust/device_ptr.h>
#include <thrust/execution_policy.h>
#include <thrust/sort.h>
#include <unordered_map>
#include "core/defines.h"
#include "core/kernel_common.h"
#include "kernel_gpu_utils.h"
#include "kernels.h"

#define EIGEN_DEFAULT_DENSE_INDEX_TYPE int
#define EIGEN_RUNTIME_NO_MALLOC
#define EIGEN_DONT_PARALLELIZE
#include <Eigen/Core>

namespace cg = cooperative_groups;

namespace SCAMP {

// Structure which manages shared memory on the GPU and automatically allocates
// appropriate segments in memory for variables used by the kernel
template <typename DATA_TYPE, typename PROFILE_DATA_TYPE, SCAMPProfileType type,
          int tile_width, int tile_height>
struct SCAMPSmem {
  __device__ SCAMPSmem(char *smem, bool compute_rows, bool compute_columns,
                       int extra_operands);
  Eigen::Map<Eigen::Array<DATA_TYPE, tile_height, 1>> df_row;
  Eigen::Map<Eigen::Array<DATA_TYPE, tile_height, 1>> dg_row;
  Eigen::Map<Eigen::Array<DATA_TYPE, tile_height, 1>> inorm_row;
  Eigen::Map<Eigen::Array<PROFILE_DATA_TYPE, tile_width, 1>> local_mp_col;
  Eigen::Map<Eigen::Array<PROFILE_DATA_TYPE, tile_height, 1>> local_mp_row;
  
  uint64_t *profile_a_length;
  uint64_t *profile_b_length;
};

template <typename DATA_TYPE, typename PROFILE_DATA_TYPE, SCAMPProfileType type,
          int tile_width, int tile_height>
__device__ SCAMPSmem<DATA_TYPE, PROFILE_DATA_TYPE, type, tile_width,
                     tile_height>::SCAMPSmem(char *smem, bool compute_rows,
                                             bool compute_columns,
                                             int extra_operands)
    : 
      df_row(nullptr),
      dg_row(nullptr),
      inorm_row(nullptr),
      local_mp_col(nullptr),
      local_mp_row(nullptr) {
  typedef decltype(df_row) TallArray;

  
  new (&df_row) TallArray((DATA_TYPE *)smem);
  smem += sizeof(DATA_TYPE) * tile_height;
  new (&dg_row) TallArray((DATA_TYPE *)smem);
  smem += sizeof(DATA_TYPE) * tile_height;
  new (&inorm_row) TallArray((DATA_TYPE *)smem);
  smem += sizeof(DATA_TYPE) * tile_height;

  if (compute_columns) {
    new (&local_mp_col) decltype(local_mp_col)((PROFILE_DATA_TYPE *)smem);
    smem += sizeof(PROFILE_DATA_TYPE) * tile_width;
  }
  if (compute_rows) {
    new (&local_mp_row) decltype(local_mp_row)((PROFILE_DATA_TYPE *)smem);
    smem += sizeof(PROFILE_DATA_TYPE) * tile_height;
  }
  if (NeedsCheckIfDone(type)) {
    profile_a_length = reinterpret_cast<uint64_t *>(smem);
    smem += sizeof(uint64_t);
    profile_b_length = reinterpret_cast<uint64_t *>(smem);
  } else {
    profile_a_length = nullptr;
    profile_b_length = nullptr;
  }
}

template <typename DATA_TYPE, typename DISTANCE_TYPE>
struct SCAMPThreadInfo {
  Eigen::Array<DATA_TYPE, DIAGS_PER_THREAD, 1> cov;
  Eigen::Array<DATA_TYPE, DIAGS_PER_THREAD, 1> dfc, dgc, inormc;
  Eigen::Array<DISTANCE_TYPE, DIAGS_PER_THREAD, 1> distc;
  Eigen::Array<unsigned int, DIAGS_PER_THREAD, 1> idxc;
  int warpln;
  int srcln;
  int updates_remaining;
  uint32_t local_row;
  uint32_t local_col;
  uint32_t global_row;
  uint32_t global_col;
};

/////////////////////////////////////////////////////////////////////////////////////
//     THESE HEADERS DEFINE COMPUTE STRATEGIES USED TO COMPUTE VARIOUS
//     PROFILE TYPES
///////////////////////////////////////////////////////////////////////////////////

#include "kernels_compute.h"
#include "kernels_smem.h"

// Computes the matrix profile given the sliding dot products for the first
// query and the precomputed data statisics
template <typename DATA_TYPE, typename PROFILE_OUTPUT_TYPE,
          typename PROFILE_DATA_TYPE, typename DISTANCE_TYPE, bool COMPUTE_ROWS,
          bool COMPUTE_COLS, SCAMPProfileType PROFILE_TYPE, int blocks_per_sm,
          int tile_height, int BLOCKSZ>
__global__ void __launch_bounds__(BLOCKSZ, blocks_per_sm)
    do_tile(SCAMPKernelInputArgs<double> args, PROFILE_OUTPUT_TYPE *profile_A,
            PROFILE_OUTPUT_TYPE *profile_B) {
  constexpr int tile_width = tile_height + BLOCKSZ * DIAGS_PER_THREAD;

  SCAMPThreadInfo<DATA_TYPE, DISTANCE_TYPE> thread_info;
  
  if (threadIdx.x == 0) {
    thread_info.srcln = 31;
  } else {
    thread_info.srcln = (threadIdx.x - 1) & 0x1f;
  }
  thread_info.warpln = (threadIdx.x) & 0x1f;
  thread_info.updates_remaining = thread_info.warpln * DIAGS_PER_THREAD + (DIAGS_PER_THREAD - 1);

  DISTANCE_TYPE init = init_dist<DISTANCE_TYPE, PROFILE_TYPE>();
  thread_info.distc = Eigen::Array<DISTANCE_TYPE, unrolled_diags, 1>::Constant(init);


  extern __shared__ char smem_raw[];

  // Wrap the shared memory in  a struct which contains handles shared memory
  // accesses
  SCAMPSmem<DATA_TYPE, PROFILE_DATA_TYPE, PROFILE_TYPE, tile_width, tile_height>
      smem(smem_raw, COMPUTE_ROWS, COMPUTE_COLS, args.opt.num_extra_operands);

  // Find the starting diagonal of the distance matrix
  const unsigned int start_diag = args.exclusion_lower +
                                  (threadIdx.x * DIAGS_PER_THREAD) +
                                  blockIdx.x * (blockDim.x * DIAGS_PER_THREAD);

  // This is the index of the meta-diagonal that this thread block will work on
  const unsigned int meta_diagonal_idx = blockIdx.x;

  // The first diagonals constitiute a trivial match between the same
  // subsequence, we must exclude these from the calculation according to
  // args.exclusion_lower
  uint32_t tile_start_col =
      meta_diagonal_idx * (BLOCKSZ * DIAGS_PER_THREAD) + args.exclusion_lower;
  uint32_t tile_start_row = 0;

  // Initialize the column and row position of the current thread
  thread_info.global_col = tile_start_col + threadIdx.x * DIAGS_PER_THREAD;
  thread_info.global_row = 0;

  // num_diags is the number of diagonals in the distance matrix, less any
  // diagonals at the end we are not computing
  const unsigned int num_diags = args.n_x - args.exclusion_upper;

  // Load the first dot product values
  for (int i = 0; i < DIAGS_PER_THREAD && thread_info.global_col + i < args.n_x;
       ++i) {
    thread_info.cov[i] = args.cov[thread_info.global_col + i];
  }

  /////////////////////////////////////
  // Main loop
  /////////////////////////////////////
  // Each threadblock finds all the distances on a 'metadiagonal'
  // We use a tiled approach for each thread block
  // The tiles are horizontal slices of the diagonal, think of a parallelogram
  // cut from a diagonal slice of the distance matrix. Each thread starts on the
  // first row and works its way down-right towards right side of the distance
  // matrix
  while (tile_start_col < args.n_x && tile_start_row < args.n_y) {
    // Initialize the next tile's shared memory
    init_smem<decltype(smem), PROFILE_DATA_TYPE, PROFILE_OUTPUT_TYPE,
              COMPUTE_ROWS, COMPUTE_COLS, tile_width, tile_height, BLOCKSZ,
              PROFILE_TYPE>(args, smem, profile_A, profile_B, tile_start_col,
                            tile_start_row);
    thread_info.local_col = threadIdx.x * DIAGS_PER_THREAD;
    thread_info.local_row = 0;

    // Start of new tile, sync so we don't have data races with shared memory
    // initializaton
    __syncthreads();

    // There are 2 pathways here, most of the time we take the fast path (top),
    // the last tile in every thread-block will take the slower path (bottom)
    if (tile_start_col + tile_width < args.n_x &&
        tile_start_row + tile_height < args.n_y &&
        start_diag + DIAGS_PER_THREAD <= num_diags) {
      // Fast Path
      while (thread_info.local_row < tile_height) {
        do_iteration_fast<PROFILE_TYPE, COMPUTE_ROWS, COMPUTE_COLS,
                          DISTANCE_TYPE>(args, thread_info, smem);
      }
    } else if (start_diag < num_diags) {
/*
      // Slow Path
      while (thread_info.global_col < args.n_x &&
             thread_info.global_row < args.n_y &&
             thread_info.local_row < tile_height) {
        do_row_edge<PROFILE_TYPE, COMPUTE_ROWS, COMPUTE_COLS, DISTANCE_TYPE>(
            args, thread_info, smem, start_diag, num_diags);
        ++thread_info.global_col;
        ++thread_info.global_row;
        ++thread_info.local_col;
        ++thread_info.local_row;
      }
*/
    }

    // After this sync, the caches will be updated with the best so far values
    // for this tile
    __syncthreads();

    // Write back our best-so-far computed for this tile to global memory
    write_back<PROFILE_TYPE, COMPUTE_COLS, COMPUTE_ROWS, BLOCKSZ, tile_width,
               tile_height>(args, smem, tile_start_col, tile_start_row,
                            args.n_x, args.n_y, profile_A, profile_B);

    // Update the tile position
    tile_start_col += tile_height;
    tile_start_row += tile_height;

    // Make sure our updates were committed before we pull in the next tile
    __threadfence_block();

    if (NeedsCheckIfDone(PROFILE_TYPE)) {
      // Copy the latest value of the profile length to shared memory
      if (threadIdx.x == 0) {
        *smem.profile_a_length = *args.profile_a_length;
        *smem.profile_b_length = *args.profile_b_length;
      }

      // Sync so that the write to shared memory is visible by all other threads
      __syncthreads();

      // If we have too many results, break this thread block out of the kernel
      // as more computation is pointless. We need to break the entire thread
      // block out at once otherwise this is undefined behavior.
      if (*smem.profile_a_length > args.max_matches_per_tile ||
          *smem.profile_b_length > args.max_matches_per_tile) {
        break;
      }
    }
  }
}

template <typename PROFILE_OUTPUT_TYPE, typename PROFILE_DATA_TYPE,
          typename DISTANCE_TYPE, SCAMPProfileType PROFILE_TYPE,
          int BLOCKSPERSM>
SCAMPError_t LaunchDoTile(SCAMPKernelInputArgs<double> args,
                          PROFILE_OUTPUT_TYPE *profile_A,
                          PROFILE_OUTPUT_TYPE *profile_B,
                          SCAMPPrecisionType fp_type, bool computing_rows,
                          bool computing_cols, uint64_t blocksz,
                          uint64_t num_blocks, uint64_t smem, hipStream_t s) {
  dim3 block(blocksz, 1, 1);
  dim3 grid(num_blocks, 1, 1);
  if (computing_rows && computing_cols) {
    constexpr bool COMPUTE_COLS = true;
    constexpr bool COMPUTE_ROWS = true;
    switch (fp_type) {
      case PRECISION_ULTRA:
      case PRECISION_DOUBLE: {
        do_tile<double, PROFILE_OUTPUT_TYPE, PROFILE_DATA_TYPE, DISTANCE_TYPE,
                COMPUTE_ROWS, COMPUTE_COLS, PROFILE_TYPE, BLOCKSPERSM,
                TILE_HEIGHT_DP, BLOCKSZ_DP>
            <<<grid, block, smem, s>>>(args, profile_A, profile_B);
        break;
      }
      case PRECISION_SINGLE: {
        do_tile<float, PROFILE_OUTPUT_TYPE, PROFILE_DATA_TYPE, DISTANCE_TYPE,
                COMPUTE_ROWS, COMPUTE_COLS, PROFILE_TYPE, BLOCKSPERSM,
                TILE_HEIGHT_SP, BLOCKSZ_SP>
            <<<grid, block, smem, s>>>(args, profile_A, profile_B);
        break;
      }
      case PRECISION_MIXED:
      default:
        return SCAMP_CUDA_ERROR;
    }
    return SCAMP_NO_ERROR;
  } else if (computing_cols) {
    constexpr bool COMPUTE_COLS = true;
    constexpr bool COMPUTE_ROWS = false;
    switch (fp_type) {
      case PRECISION_ULTRA:
      case PRECISION_DOUBLE: {
        do_tile<double, PROFILE_OUTPUT_TYPE, PROFILE_DATA_TYPE, DISTANCE_TYPE,
                COMPUTE_ROWS, COMPUTE_COLS, PROFILE_TYPE, BLOCKSPERSM,
                TILE_HEIGHT_DP, BLOCKSZ_DP>
            <<<grid, block, smem, s>>>(args, profile_A, profile_B);
        break;
      }
      case PRECISION_SINGLE: {
        do_tile<float, PROFILE_OUTPUT_TYPE, PROFILE_DATA_TYPE, DISTANCE_TYPE,
                COMPUTE_ROWS, COMPUTE_COLS, PROFILE_TYPE, BLOCKSPERSM,
                TILE_HEIGHT_SP, BLOCKSZ_SP>
            <<<grid, block, smem, s>>>(args, profile_A, profile_B);
        break;
      }
      case PRECISION_MIXED:
      default:
        return SCAMP_CUDA_ERROR;
    }
  } else if (computing_rows) {
    constexpr bool COMPUTE_COLS = false;
    constexpr bool COMPUTE_ROWS = true;
    switch (fp_type) {
      case PRECISION_ULTRA:
      case PRECISION_DOUBLE: {
        do_tile<double, PROFILE_OUTPUT_TYPE, PROFILE_DATA_TYPE, DISTANCE_TYPE,
                COMPUTE_ROWS, COMPUTE_COLS, PROFILE_TYPE, BLOCKSPERSM,
                TILE_HEIGHT_DP, BLOCKSZ_DP>
            <<<grid, block, smem, s>>>(args, profile_A, profile_B);
        break;
      }
      case PRECISION_SINGLE: {
        do_tile<float, PROFILE_OUTPUT_TYPE, PROFILE_DATA_TYPE, DISTANCE_TYPE,
                COMPUTE_ROWS, COMPUTE_COLS, PROFILE_TYPE, BLOCKSPERSM,
                TILE_HEIGHT_SP, BLOCKSZ_SP>
            <<<grid, block, smem, s>>>(args, profile_A, profile_B);
        break;
      }
      case PRECISION_MIXED:
      default:
        return SCAMP_CUDA_ERROR;
    }
  }
  gpuErrchk(hipPeekAtLastError());
  return SCAMP_NO_ERROR;
}

SCAMPError_t compute_gpu_resources_and_launch(SCAMPKernelInputArgs<double> args,
                                              Tile *t, void *profile_a,
                                              void *profile_b, bool do_rows,
                                              bool do_cols) {
  int exclusion_total = args.exclusion_lower + args.exclusion_upper;
  uint64_t blocksz = get_blocksz(t->info()->fp_type);
  uint64_t num_workers = ceil((args.n_x - exclusion_total) /
                              static_cast<double>(DIAGS_PER_THREAD));
  uint64_t num_blocks = ceil(num_workers / static_cast<double>(blocksz));
  uint64_t smem = get_smem(t->info(), blocksz);
  if (!t->info()->silent_mode) {
    std::cout << "Launching " << num_blocks << " thread blocks of size "
              << blocksz << " with a total of " << smem
              << " bytes of shared memory per block." << std::endl;
  }
  if (exclusion_total < args.n_x) {
    switch (t->info()->profile_type) {
      case PROFILE_TYPE_SUM_THRESH:
        return LaunchDoTile<double, double, double, PROFILE_TYPE_SUM_THRESH,
                            BLOCKSPERSM>(
            args, reinterpret_cast<double *>(profile_a),
            reinterpret_cast<double *>(profile_b), t->info()->fp_type, do_rows,
            do_cols, blocksz, num_blocks, smem, t->get_stream());
      case PROFILE_TYPE_1NN_INDEX:
        return LaunchDoTile<uint64_t, uint64_t, float, PROFILE_TYPE_1NN_INDEX,
                            BLOCKSPERSM>(
            args, reinterpret_cast<uint64_t *>(profile_a),
            reinterpret_cast<uint64_t *>(profile_b), t->info()->fp_type,
            do_rows, do_cols, blocksz, num_blocks, smem, t->get_stream());
      case PROFILE_TYPE_1NN:
        return LaunchDoTile<float, float, float, PROFILE_TYPE_1NN, BLOCKSPERSM>(
            args, reinterpret_cast<float *>(profile_a),
            reinterpret_cast<float *>(profile_b), t->info()->fp_type, do_rows,
            do_cols, blocksz, num_blocks, smem, t->get_stream());
      case PROFILE_TYPE_APPROX_ALL_NEIGHBORS:
        return LaunchDoTile<SCAMPmatch, uint64_t, float,
                            PROFILE_TYPE_APPROX_ALL_NEIGHBORS, BLOCKSPERSM>(
            args, reinterpret_cast<SCAMPmatch *>(profile_a),
            reinterpret_cast<SCAMPmatch *>(profile_b), t->info()->fp_type,
            do_rows, do_cols, blocksz, num_blocks, smem, t->get_stream());
      case PROFILE_TYPE_MATRIX_SUMMARY:
        return LaunchDoTile<float, uint64_t, float, PROFILE_TYPE_MATRIX_SUMMARY,
                            BLOCKSPERSM>(
            args, reinterpret_cast<float *>(profile_a),
            reinterpret_cast<float *>(profile_b), t->info()->fp_type, do_rows,
            do_cols, blocksz, num_blocks, smem, t->get_stream());
      default:
        return SCAMP_FUNCTIONALITY_UNIMPLEMENTED;
    }
  }
  return SCAMP_NO_ERROR;
}

SCAMPError_t gpu_kernel_self_join_upper(Tile *t) {
  SCAMPKernelInputArgs<double> tile_args(t, false, false);
  return compute_gpu_resources_and_launch(
      tile_args, t, t->profile_a(), t->profile_b(), t->info()->computing_rows,
      t->info()->computing_cols);
}

SCAMPError_t gpu_kernel_self_join_lower(Tile *t) {
  SCAMPKernelInputArgs<double> tile_args(t, true, false);
  return compute_gpu_resources_and_launch(
      tile_args, t, t->profile_b(), t->profile_a(), t->info()->computing_cols,
      t->info()->computing_rows);
}

SCAMPError_t gpu_kernel_ab_join_upper(Tile *t) {
  SCAMPKernelInputArgs<double> tile_args(t, false, true);
  return compute_gpu_resources_and_launch(
      tile_args, t, t->profile_a(), t->profile_b(), t->info()->computing_rows,
      t->info()->computing_cols);
}

SCAMPError_t gpu_kernel_ab_join_lower(Tile *t) {
  SCAMPKernelInputArgs<double> tile_args(t, true, true);
  return compute_gpu_resources_and_launch(
      tile_args, t, t->profile_b(), t->profile_a(), t->info()->computing_cols,
      t->info()->computing_rows);
}

void match_gpu_sort(SCAMPmatch *matches, int64_t len, hipStream_t stream) {
  thrust::device_ptr<SCAMPmatch> ptr = thrust::device_pointer_cast(matches);
  thrust::sort(thrust::cuda::par.on(stream), ptr, ptr + len);
}

}  // namespace SCAMP
